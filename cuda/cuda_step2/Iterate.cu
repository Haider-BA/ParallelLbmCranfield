#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>                       // CUDA
#include <stdio.h>                      // printf();
#include <math.h>                       // need to compile with -lm
#include <stdlib.h>                     // for calloc();
#include <stdbool.h>                    // Include for bool type variables!
#include <string.h>                     // String operations
#include <time.h>                       // time functions
#include "include/ShellFunctions.h"     // For convenience
#include "include/FilesReading.h"       // For reading files
#include "include/FilesWriting.h"       // For writing files e.g. tecplot
#include "include/CellFunctions.h"      // For cell modifications
#include "include/ComputeResiduals.h"   // Residuals

#define threads 256

__constant__ int InletProfile_d[1];
__constant__ int NumNodes_d[1];
__constant__ int NumConn_d[1];
__constant__ int CurvedBoundaries_d[1];
__constant__ int OutletProfile_d[1];
__constant__ int CalculateDragLift_d[1];
__constant__ int cx_d[9]; 
__constant__ int cy_d[9];
__constant__ int width_d[1];
__constant__ int height_d[1];
__constant__ int c_d[9];
__constant__ int opp_d[9];
__constant__ float Delta_d[1];
__constant__ float w_d[9];
__constant__ float Qlat_d[9];
__constant__ float omega_d[1];
__constant__ float omegaA_d[1];
__constant__ float rho_ini_d[1];
__constant__ float Uavg_d[1];
__constant__ float Vavg_d[1];
__constant__ float MinInletCoordY_d[1];
__constant__ float MaxInletCoordY_d[1];
__constant__ float tm_d[81];
__constant__ float stmiv_d[81];

  ////////////////////////////////////////////////////
  ///////////////// KERNEL FCTS //////////////////////
  ////////////////////////////////////////////////////

  ////////////////////////////////////////////////////
  /////////////// Initialization1 ////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_init1(int *Fluid_d, int *ID_d, int *Boundary_d, int *BoundaryID_d, float *CoordX_d, float *CoordY_d,
                          int *BC_ID_d, float *Q_d, float *U_d, float *V_d, float *Rho_d,
                         int *Nodes0_d, int *Nodes1_d, float *Nodes2_d, float *Nodes3_d, int *Nodes4_d,
                         int *BCconn0_d, int *BCconn1_d, int *BCconn2_d, int *BCconn3_d, float *BCconn4_d,
                         float *BCconn5_d, int *BCconn6_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  if (ind<((*width_d)*(*height_d)))
  {

    ID_d[ind] = ind;

    CoordX_d[ind] = Nodes2_d[ind];
    CoordY_d[ind] = Nodes3_d[ind];
  
    // CHECK FLUID OR NOT
    Fluid_d[ind] = Nodes4_d[ind];

    U_d[ind] = 0;
    V_d[ind] = 0;
    Rho_d[ind] = *rho_ini_d;

    Boundary_d[ind] = 0;
    BoundaryID_d[ind] = 0;
  }

  if (ind<(9*(*width_d)*(*height_d)))
  {
      BC_ID_d[ind]= 0  ; // IT IS NOT BOUNDARY LATTICE
      Q_d[ind]    = 0.5; 
  }
}


  ////////////////////////////////////////////////////
  /////////////// Initialization2 ////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_init2(int *Corner_d, int *Boundary_d, float *CoordY_d, int *StreamLattice_d, int *BC_ID_d, float *U_d, float *V_d, float *Uo_d, float *Vo_d,
                         int *Nodes0_d, int *Nodes1_d, float *Nodes2_d, float *Nodes3_d, int *Nodes4_d,
                         int *BCconn0_d, int *BCconn1_d, int *BCconn2_d, int *BCconn3_d, float *BCconn4_d,
                         float *BCconn5_d, int *BCconn6_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ms=(*width_d)*(*height_d);

  int k;

  if (ind<((*width_d)*(*height_d)))
  {
    // not in the corner
    Corner_d[ind]=0;
  }

  if (ind<(9*(*width_d)*(*height_d)))
  {

    if (BC_ID_d[ind]!=0)
    {
      if (Boundary_d[ind_s]==0)// if Boundary condition in the node is 0 it becomes equal to the BC of the lattice direction
      {
        Boundary_d[ind_s]=BC_ID_d[ind];
      }
      else
      {// if in the same node there are lattice directions with different BC (corners) the BC of the node is WALL (assuming that it's impossibe to findoutlet and inlet together)
        if (Boundary_d[ind_s]<BC_ID_d[ind])
        {
          Boundary_d[ind_s]=1;
          Corner_d[ind_s]=1;
        }
        if (Boundary_d[ind_s]>BC_ID_d[ind])
        {
          Boundary_d[ind_s]=1;
          Corner_d[ind_s]=1;
        }
      }
    }


    // BC ON CORNERS IS WALL!!!! (this operation is useful for wall condition, which checks the single direction)
    if (Corner_d[ind_s]==1)
    {
      if (BC_ID_d[ind_s+(*width_d)*(*height_d)]!=0 && BC_ID_d[ind_s+2*(*width_d)*(*height_d)]!=0)
      {
          BC_ID_d[ind_s+5*(*width_d)*(*height_d)]=1;
      }
      if (BC_ID_d[ind_s+(*width_d)*(*height_d)]!=0 && BC_ID_d[ind_s+4*(*width_d)*(*height_d)]!=0)
      {
          BC_ID_d[ind_s+8*(*width_d)*(*height_d)]=1;
      }
      if (BC_ID_d[ind_s+2*(*width_d)*(*height_d)]!=0 && BC_ID_d[ind_s+3*(*width_d)*(*height_d)]!=0)
      {
          BC_ID_d[ind_s+6*(*width_d)*(*height_d)]=1;
      }
      if (BC_ID_d[ind_s+3*(*width_d)*(*height_d)]!=0 && BC_ID_d[ind_s+4*(*width_d)*(*height_d)]!=0)
      {
          BC_ID_d[ind_s+7*(*width_d)*(*height_d)]=1;
      }
    }

    // INITIALIZE STREAMING (STREAM EVERYWHERE)
    StreamLattice_d[ind] = 1;

  }

  if (ind<(9*(*width_d)*(*height_d)))
  {
    // DON'T STREAM FROM OUTSIDE OF THE DOMAIN
    for(k=0;k<9;k++)
    {
      if (BC_ID_d[ind_s+k*ms]!=0)
      {
        StreamLattice_d[ind_s+opp_d[k]]= 0 ;
      }
    }
  }

  if (ind<((*width_d)*(*height_d)))
  {  
  // INLET VELOCITY
    switch(*InletProfile_d)
    {
      case 1:
        Uo_d[ind_s] = 4*1.5*(*Uavg_d)*(CoordY_d[ind_s]-(*MinInletCoordY_d))*(((*MaxInletCoordY_d)-
                 (*MinInletCoordY_d))-(CoordY_d[ind_s]-(*MinInletCoordY_d)))/(((*MaxInletCoordY_d)-
                 (*MinInletCoordY_d))*((*MaxInletCoordY_d)-(*MinInletCoordY_d)));

        Vo_d[ind_s] = *Vavg_d;
      break;
      case 2:
        Uo_d[ind_s] = *Uavg_d;
        Vo_d[ind_s] = *Vavg_d;
      break;
      case 3:
        Uo_d[ind_s]= 0;
        Vo_d[ind_s] = 0;
      break;

    }
    U_d[ind_s] = Uo_d[ind_s];
    V_d[ind_s] = Vo_d[ind_s];

  } 


}


  ////////////////////////////////////////////////////
  ////////////////// BGKW model //////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_bgk(int* Fluid_d, float* Feq_d, float* Rho_d, float* U_d, float* V_d, float* METAF_d, float* F_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ind_c = (int)(ind/((*width_d)*(*height_d)));

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
      // Collision
      Feq_d[ind]  = Rho_d[ind_s]*w_d[ind_c]*( 1.0 + 3.0*(U_d[ind_s]*cx_d[ind_c] + V_d[ind_s]*cy_d[ind_c])
                + 4.5*(U_d[ind_s]*cx_d[ind_c] + V_d[ind_s]*cy_d[ind_c])
                * (U_d[ind_s]*cx_d[ind_c] + V_d[ind_s]*cy_d[ind_c])
                - 1.5*(U_d[ind_s] * U_d[ind_s] + V_d[ind_s]* V_d[ind_s]) );
      METAF_d[ind] = (*omega_d)*Feq_d[ind]+(1.0-(*omega_d))*F_d[ind];
    }
  }
}



  ////////////////////////////////////////////////////
  ////////////////// TRT model1 //////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_trt1(int* Fluid_d, float* Feq_d, float* Rho_d, float* U_d, float* V_d)
{
  
  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ind_c = (int)(ind/((*width_d)*(*height_d)));

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
      Feq_d[ind] = Rho_d[ind_s]*w_d[ind_c] * (1.0+3.0*(U_d[ind_s]*cx_d[ind_c]+V_d[ind_s]*cy_d[ind_c])
               + 4.5 *(U_d[ind_s]*cx_d[ind_c]+V_d[ind_s]*cy_d[ind_c]) * (U_d[ind_s]*cx_d[ind_c]+V_d[ind_s]*cy_d[ind_c])
               - 0.5*(U_d[ind_s]*U_d[ind_s]+V_d[ind_s]*V_d[ind_s]));
    }
  }
}

  ////////////////////////////////////////////////////
  ////////////////// TRT model2 //////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_trt2(int* Fluid_d, float* Feq_d, float* F_d, float* METAF_d)
{
  
  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ind_c = (int)(ind/((*width_d)*(*height_d)));

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
      METAF_d[ind] = F_d[ind] 
                   - ((0.5*(F_d[ind]   + F_d[ind_s+opp_d[ind_c]]))
                   - (0.5* (Feq_d[ind] + Feq_d[ind_s+opp_d[ind_c]])))*(*omega_d)
                   - ((0.5*(F_d[ind]   - F_d[ind_s+opp_d[ind_c]]))
                   - (0.5* (Feq_d[ind] - Feq_d[ind_s+opp_d[ind_c]])))*(*omegaA_d);

    }
  }
}


  ////////////////////////////////////////////////////
  ////////////////// MRT model1 //////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_mrt1(int* Fluid_d, float* Rho_d, float* U_d, float* V_d, float* F_d, float* fmeq_d, float* fmom_d)
{
  
  int l;

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;
  
  int ms=(*width_d)*(*height_d);
  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ind_c = (int)(ind/((*width_d)*(*height_d)));

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {

      fmeq_d[ind_s]      = Rho_d[ind_s];
      fmeq_d[ind_s+1*ms] = Rho_d[ind_s]*(-2.0+3.0*Rho_d[ind_s]*
                             (U_d[ind_s]*U_d[ind_s]+V_d[ind_s]*V_d[ind_s]));
      fmeq_d[ind_s+2*ms] = Rho_d[ind_s]*(1.0-3.0*Rho_d[ind_s]*
                             (U_d[ind_s]*U_d[ind_s]+V_d[ind_s]*V_d[ind_s]));
      fmeq_d[ind_s+3*ms] = Rho_d[ind_s]*U_d[ind_s];
      fmeq_d[ind_s+4*ms] =-Rho_d[ind_s]*U_d[ind_s];
      fmeq_d[ind_s+5*ms] = Rho_d[ind_s]*V_d[ind_s];
      fmeq_d[ind_s+6*ms] =-Rho_d[ind_s]*V_d[ind_s];
      fmeq_d[ind_s+7*ms] = Rho_d[ind_s]*(U_d[ind_s]*U_d[ind_s]-
                             V_d[ind_s]*V_d[ind_s]);
      fmeq_d[ind_s+8*ms] = Rho_d[ind_s]*U_d[ind_s]*V_d[ind_s];
  

      fmom_d[ind]=0;
      for (l=0; l<9;l++)
      fmom_d[ind]=fmom_d[ind] + tm_d[ind_c*9+l]*F_d[ind_s+l*ms];

    }
  }
}


  ////////////////////////////////////////////////////
  ////////////////// MRT model2 //////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_mrt2(int* Fluid_d, float* sumb_d, float* fmom_d, float* fmeq_d, float* METAF_d, float* F_d)
{
  
  int l;

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;
  
  int ms=(*width_d)*(*height_d);
  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ind_c = (int)(ind/((*width_d)*(*height_d)));

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
      sumb_d[ind] = 0;

        for (l=0; l<9;l++)
          sumb_d[ind] = sumb_d[ind] + (stmiv_d[ind_c*9+l]*(fmom_d[ind_s+l*ms]-fmeq_d[ind_s+l*ms]));

        METAF_d[ind] = F_d[ind] - sumb_d[ind];

    }
  }
}





  ////////////////////////////////////////////////////
  //////////// UPDATE DISTR. FCT. ////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_update_f(int* Fluid_d, float* F_d, float* METAF_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );


  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
      // Update F
      F_d[ind] = METAF_d[ind];
    }
  }
}


  ////////////////////////////////////////////////////
  ///////////////// STREAMING ////////////////////////
  ////////////////////////////////////////////////////


__global__ void gpu_streaming(int* Fluid_d, int* StreamLattice_d, float* F_d, float* METAF_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ind_c = (int)(ind/((*width_d)*(*height_d)));

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
      // STREAMING 
      if ( (StreamLattice_d[ind]) == 1 )
      {
        F_d[ind] = METAF_d[ind+c_d[ind_c]];
      }
    }
  }
}



  ////////////////////////////////////////////////////
  //////////////// boundaries1 ///////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_boundaries1(int* Fluid_d, int* Boundary_d, int* BC_ID_d, float* F_d, float* Uo_d, float* Vo_d, int* Corner_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ms=(*width_d)*(*height_d);

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
    
      //inlet BC 
      if (Boundary_d[ind_s]==2) // if inlet boundary
      {
        
        /* if (_d[ind_s+ms].BC_ID==2 && _d[ind_s].Corner!=1)
        {
          // corner treatment has to be work out 
        }*/

        // inlet on the top (to be completed with the y velocity)
        if (BC_ID_d[ind_s+2*ms]==2 && Corner_d[ind_s]!=1)
        {

          F_d[ind_s+4*ms] = F_d[ind_s+2*ms];

          F_d[ind_s+8*ms] = F_d[ind_s+6*ms]
                                                    + (F_d[ind_s] + F_d[ind_s+1*ms]
                                                    + F_d[ind_s+3*ms]
                                                    + 2*(F_d[ind_s+2*ms]
                                                    + F_d[ind_s+6*ms]
                                                    + F_d[ind_s+5*ms]))*Uo_d[ind_s]/6.0;

          F_d[ind_s+7*ms] = F_d[ind_s+5*ms]
                                                    - (F_d[ind_s] + F_d[ind_s+1*ms]
                                                    + F_d[ind_s+3*ms]
                                                    + 2*(F_d[ind_s+2*ms]
                                                    + F_d[ind_s+6*ms]
                                                    + F_d[ind_s+5*ms]))*Uo_d[ind_s]/6.0;
        }

        // inlet on the left (to be completed with the x velocity)
        if (BC_ID_d[ind_s+3*ms]==2 && Corner_d[ind_s]!=1)
        {

            F_d[ind_s+1*ms] = F_d[ind_s+3*ms]
                              + 2*((F_d[ind_s]+F_d[ind_s+2*ms]
                              + F_d[ind_s+4*ms]
                              + 2.*(F_d[ind_s+3*ms]
                              + F_d[ind_s+6*ms]
                              + F_d[ind_s+7*ms]))
                              / (1.0-Uo_d[ind_s]))*Uo_d[ind_s]/3;

            F_d[ind_s+5*ms] = F_d[ind_s+7*ms]
                              +  ((F_d[ind_s]+F_d[ind_s+2*ms]
                              + F_d[ind_s+4*ms]
                              + 2.*(F_d[ind_s+3*ms]
                              + F_d[ind_s+6*ms] 
                              + F_d[ind_s+7*ms]))
                              / (1.0-Uo_d[ind_s]))*Uo_d[ind_s]/6;

            F_d[ind_s+8*ms] = F_d[ind_s+6*ms]
                              +  ((F_d[ind_s]+ F_d[ind_s+2*ms]
                              +F_d[ind_s+4*ms] 
                              + 2.*(F_d[ind_s+3*ms]
                              +F_d[ind_s+6*ms]
                              + F_d[ind_s+7*ms]))
                              / (1.0-Uo_d[ind_s]))*Uo_d[ind_s]/6;
        }
    
        /*if (_d[ind_s+4*ms].BC_ID==2 && _d[ind_s].Corner!=1)
        {
          // corner treatment has to be work out 
        }*/
      }
    }
  }  
}



  ////////////////////////////////////////////////////
  //////////////// boundaries2 ///////////////////////
  ////////////////////////////////////////////////////


__global__ void gpu_boundaries2(int* Fluid_d, float* Fneighbours_d, float* METAF_d, int* BC_ID_d, float* Q_d, float* F_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ind_c = (int)(ind/((*width_d)*(*height_d)));

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {


      // WALL
      switch(*CurvedBoundaries_d)
      {
        // curved boundaries
      case 1:
 

          Fneighbours_d[ind]=METAF_d[ind+c_d[ind_c]];

          //CURVED WALL BC 

          if (BC_ID_d[ind]==1) // if wall
          {
            if (Q_d[ind]<0.5) // if the distance from the boundary is less than 0.5?
            { 
              F_d[ind_s+opp_d[ind_c]] = 2*Q_d[ind]*METAF_d[ind]
                                +(1-2*Q_d[ind])*Fneighbours_d[ind];
            }
            else
            {
              F_d[ind_s+opp_d[ind_c]] = METAF_d[ind]/2/Q_d[ind]
                                + (2*Q_d[ind]-1)/(2*Q_d[ind])
                                * METAF_d[ind_s+opp_d[ind_c]];
            }
          }

      break;

        // bounceback boundaries
      case 2:         
        //WALL BC (half-way bounceback)
        if (BC_ID_d[ind]==1) // if wall boundary
        {
          F_d[ind_s+opp_d[ind_c]] = F_d[ind];
        }
      break;
      }
    }
  }
}



  ////////////////////////////////////////////////////
  //////////////// boundaries3 ///////////////////////
  ////////////////////////////////////////////////////

__global__ void gpu_boundaries3(int* Fluid_d, int* BC_ID_d, float* F_d, float* Uo_d, float* Vo_d, int* Corner_d)
{

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ind_s = ind - ((*width_d)*(*height_d)) *( (int)(ind/((*width_d)*(*height_d))) );

  int ms=(*width_d)*(*height_d);

  if (ind<(9*(*width_d)*(*height_d)))
  {
    if (Fluid_d[ind_s]==1)
    {
      // OUTLET
      switch(*OutletProfile_d)
      {
        // set profile in outlet
        case 1:
        //outlet BC 
              if (BC_ID_d[ind_s+ms]==3) // if outlet boundary on the right side of the domain
          {

            F_d[ind_s+3*ms] = F_d[ind_s+1*ms]-2*((F_d[ind_s]
                                        + F_d[ind_s+2*ms]+F_d[ind_s+4*ms]
                                        + 2.*(F_d[ind_s+1*ms]+F_d[ind_s+5*ms]
                                        + F_d[ind_s+8*ms]))/(1-Uo_d[ind_s]))*Uo_d[ind_s]/3;

            F_d[ind_s+7*ms] = F_d[ind_s+5*ms]-((F_d[ind_s]
                                        + F_d[ind_s+2*ms]+F_d[ind_s+4*ms]
                                        + 2.*(F_d[ind_s+4*ms]+F_d[ind_s+5*ms]
                                        + F_d[ind_s+8*ms]))/(1-Uo_d[ind_s]))*Uo_d[ind_s]/6;

            F_d[ind_s+6*ms] = F_d[ind_s+8*ms]-((F_d[ind_s]
                                        + F_d[ind_s+2*ms]+F_d[ind_s+4*ms]
                                        + 2.*(F_d[ind_s+1*ms]+F_d[ind_s+5*ms]
                                        + F_d[ind_s+8*ms]))/(1-Uo_d[ind_s]))*Uo_d[ind_s]/6;
          }
          if (BC_ID_d[ind_s+2*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
          if (BC_ID_d[ind_s+3*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
          if (BC_ID_d[ind_s+4*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
        break;

         // OPEN BOUNDARY
        case 2 :
          if (BC_ID_d[ind_s+ms]==3)
          {
            F_d[ind_s+ms] = 2*F_d[ind_s+ms-1]-F_d[ind_s+ms-2];
            F_d[ind_s+5*ms] = 2*F_d[ind_s+5*ms-1]-F_d[ind_s+5*ms-2];
            F_d[ind_s+8*ms] = 2*F_d[ind_s+8*ms-1]-F_d[ind_s+8*ms-2];
          }
          if (BC_ID_d[ind+2*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
          if (BC_ID_d[ind+3*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
          if (BC_ID_d[ind+4*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
        break;
        // first order outlet
        case 3 :
          if (BC_ID_d[ind_s+ms]==3)
          {
            F_d[ind_s+ms] = F_d[ind_s-1+ms];
            F_d[ind_s+5*ms] = F_d[ind_s-1+5*ms];
            F_d[ind_s+8*ms] = F_d[ind_s-1+8*ms];
          }
          if (BC_ID_d[ind+2*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
          if (BC_ID_d[ind+3*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
          if (BC_ID_d[ind+4*ms]==3 && Corner_d[ind_s]!=1)
          {
          // FILL!!
          }
        break;
      }

    }
  }
}




  ////////////////////////////////////////////////////
  //////////// UPDATE MACROSCOPIC ////////////////////
  ////////////////////////////////////////////////////



__global__ void gpu_update_macro(int* Fluid_d, float* Rho_d, float* U_d, float* V_d,
                                int* BC_ID_d, int* BoundaryID_d, float* DragF_d,
                                float* LiftF_d, float* CoordX_d, float* CoordY_d, float* F_d)
{
  
  int k;

  int bidx=blockIdx.x;
  int tidx=threadIdx.x;

  int ind = tidx + bidx*blockDim.x;

  int ms = (*width_d)*(*height_d);

  if (ind<((*width_d)*(*height_d)))
  {
    if (Fluid_d[ind]==1)
    {
      // Update macroscopic
      Rho_d[ind]=0;
      U_d[ind]=0;
      V_d[ind]=0;
      for (k=0; k<9; k++)
      {        
        Rho_d[ind] = Rho_d[ind] + F_d[ind+k*ms];
        U_d[ind] = U_d[ind] + F_d[ind+k*ms]*cx_d[k];
        V_d[ind]= V_d[ind] + F_d[ind+k*ms]*cy_d[k];
      }

      U_d[ind] = U_d[ind] / Rho_d[ind];
      V_d[ind] = V_d[ind] / Rho_d[ind];

      if (BC_ID_d[ind+ms]==3) // for outlet on the right
      {
        V_d[ind]=0.0;
      }

      //   DRAG/LIFT FORCE
      if (*CalculateDragLift_d != 0 && BoundaryID_d[ind]==*CalculateDragLift_d)
      {
         DragF_d[ind] = 0.33333333*Rho_d[ind]*(20-CoordX_d[ind])*0.2;
         LiftF_d[ind] = 0.33333333*Rho_d[ind]*(20-CoordY_d[ind])*0.2;
      }
    }
  }
}




  ////////////////////////////////////////////////////
  ///////////////// SOLVER ///////////////////////////
  ////////////////////////////////////////////////////




void Iteration(char* NodeDataFile, char* BCconnectorDataFile,
               float Uavg,         float Vavg,         float rho_ini, float Viscosity,
               int InletProfile,   int CollisionModel, int CurvedBoundaries,
               int OutletProfile,  int Iterations,     int AutosaveAfter,
               int AutosaveEvery, int postproc_prog,   int CalculateDragLift)
{

  ////////////////////////////////////////////////////
  ///////////////////// Declare //////////////////////
  ////////////////////////////////////////////////////
  
  int bx; // blocks in X direction


  // Time measurement: declaration, begin
  clock_t tStart = clock();

  FILE* resid_file;                       // file for residuals
  FILE* log_file;                         // file for log
  FILE* TimeMeasurementFile;              // file for time measurement results
  // char IterOutputFile[50];                // write results to this file after the iterations
  char AutosaveOutputFile[50];            // autosave filename
  char OutputFile[50];                    // initial data will be written to this file
  char FinalOutputFile[50];               // final data will be written to this file
  char logFile[] = "Results/logFile.log"; // path of the .log file
  int i, j, A, B, ind, ind9, iter = 0;                  //variables for loops

  // Variables for residuals
  float *Residuals;

  float Qlat[9]={0.,1.,1.,1.,1.,sqrt(2),sqrt(2),sqrt(2),sqrt(2)};

  int AutosaveI = 1;      // autosave i variable, will be incremented after every autosave
  int* ppp;               // pointer of the postproc_prog variable
  int *NumNodes,*NumConn; // This will store the number of lines of the read files
  float *Delta;           // grid spacing
  int *n,*m;              // number of nodes in the x and y directions
  float *MaxInletCoordY; // maximum inlet coordinate in y
  float *MinInletCoordY; // minimum inlet coordinate in y
  int *NumInletNodes;     // number of inlet nodes
  int *Nodes0, *Nodes1, *Nodes4, *BCconn0, *BCconn1, *BCconn2, *BCconn3, *BCconn6; // vectors for the nodes and connections
  float *Nodes2, *Nodes3,*BCconn4, *BCconn5; // vectors for the nodes and connections
  float Omega, OmegaA;   // collision frequency from the viscosity


  float tInitialization = 0.0;  // Time measurement of Initialization
  float tIteration      = 0.0;  // Time measurement of Iteration
  float tCollision      = 0.0;  // Time measurement of Collision
  float tUpdateF        = 0.0;  // Time measurement of UpdateF
  float tStreaming      = 0.0;  // Time measurement of Streaming
  float tBoundaries     = 0.0;  // Time measurement of Boundaries
  float tUpdateMacro    = 0.0;  // Time measurement of Update Macroscopic vars
  float tResiduals      = 0.0;  // Time measurement of calculating residuals
  float tWriting        = 0.0;  // Time measurement of writing data

  clock_t tInstant1, tInstant2; // Time measurement points, universal
  clock_t tIterStart, tIterEnd; // Time measurement points: main loop

  // cuda time measurement variables
  hipEvent_t start, stop;
  float cudatime;
  hipEventCreate(&start);
  hipEventCreate(&stop);  
 

  ////////////////////////////////////////////////////
  ////////////////// Declare GPU /////////////////////
  ////////////////////////////////////////////////////

  // scalar on host
  float *omega_h, *omegaA_h;
  int *OutletProfile_h;
  int *CurvedBoundaries_h;
  int *CalculateDragLift_h;

  // vectors on device (variables
  int *Nodes0_d, *Nodes1_d, *Nodes4_d, *BCconn0_d, *BCconn1_d, *BCconn2_d, *BCconn3_d, *BCconn6_d; // vectors for the nodes and connections
  float *Nodes2_d, *Nodes3_d, *BCconn4_d, *BCconn5_d; // vectors for the nodes and connections
  


  ////////////////////////////////////////////////////
  //////////////////// Allocate //////////////////////
  ////////////////////////////////////////////////////

  // allocate residuals
  Residuals = Create1DArrayFloat(4); 

  // allocate mesh properties  
  Delta          = Create1DArrayFloat(1);
  m              = Create1DArrayInt(1);
  n              = Create1DArrayInt(1);
  NumNodes       = Create1DArrayInt(1);
  NumConn        = Create1DArrayInt(1);
  MaxInletCoordY = Create1DArrayFloat(1);
  MinInletCoordY = Create1DArrayFloat(1);
  NumInletNodes  = Create1DArrayInt(1);


  // D2Q9 Variables of the lattice
  float* w = Create1DArrayFloat(9); // weight values for the directions
  int*   c_h = Create1DArrayInt(9);    // x coordinate of the discrete lattice directions
  int*   cx = Create1DArrayInt(9);    // x coordinate of the discrete lattice directions
  int*   cy = Create1DArrayInt(9);    // y coordinate of the discrete lattice directions
  int*  opp = Create1DArrayInt(9);    // opposite vector




  ////////////////////////////////////////////////////
  ///////////////////// Read data ////////////////////
  ////////////////////////////////////////////////////

  ReadNodLines(NodeDataFile, NumNodes);

  // allocate vectors for nodes
  Nodes0 = (int *)calloc(*(NumNodes),sizeof(int));
  Nodes1 = (int *)calloc(*(NumNodes),sizeof(int));
  Nodes2 = (float *)calloc(*(NumNodes),sizeof(float));
  Nodes3 = (float *)calloc(*(NumNodes),sizeof(float));
  Nodes4 = (int *)calloc(*(NumNodes),sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL(NumNodes_d), NumNodes, sizeof(int));

  hipMalloc((void**)&Nodes0_d, (*NumNodes) * sizeof(int));
  hipMalloc((void**)&Nodes1_d, (*NumNodes) * sizeof(int));
  hipMalloc((void**)&Nodes2_d, (*NumNodes) * sizeof(float));
  hipMalloc((void**)&Nodes3_d, (*NumNodes) * sizeof(float));
  hipMalloc((void**)&Nodes4_d, (*NumNodes) * sizeof(int));



  // Read Node data
  ReadNodes(NodeDataFile, NumNodes, Nodes0, Nodes1, Nodes2, Nodes3, Nodes4);

  ReadBCconLines(BCconnectorDataFile, NumConn);

  // allocate vectors for connectors
  BCconn0 = (int *)calloc(*(NumConn),sizeof(int));
  BCconn1 = (int *)calloc(*(NumConn),sizeof(int));
  BCconn2 = (int *)calloc(*(NumConn),sizeof(int));
  BCconn3 = (int *)calloc(*(NumConn),sizeof(int));
  BCconn4 = (float *)calloc(*(NumConn),sizeof(float));
  BCconn5 = (float *)calloc(*(NumConn),sizeof(float));
  BCconn6 = (int *)calloc(*(NumConn),sizeof(int)); 

  hipMemcpyToSymbol(HIP_SYMBOL(NumConn_d), NumConn, sizeof(int));

  hipMalloc((void**)&BCconn0_d, (*NumConn) * sizeof(int));
  hipMalloc((void**)&BCconn1_d, (*NumConn) * sizeof(int));
  hipMalloc((void**)&BCconn2_d, (*NumConn) * sizeof(int));
  hipMalloc((void**)&BCconn3_d, (*NumConn) * sizeof(int));
  hipMalloc((void**)&BCconn4_d, (*NumConn) * sizeof(float));
  hipMalloc((void**)&BCconn5_d, (*NumConn) * sizeof(float));
  hipMalloc((void**)&BCconn6_d, (*NumConn) * sizeof(int));

  // Read BCconn data
  ReadBCconn(BCconnectorDataFile, NumConn, BCconn0, BCconn1, BCconn2, BCconn3, BCconn4, BCconn5, BCconn6); 

  ////////////////////////////////////////////////////
  /////////////// Compute constants //////////////////
  ////////////////////////////////////////////////////

  CompDataNode(Delta, m,  n, Nodes0, Nodes1, Nodes2, Nodes3, Nodes4, NumNodes);

  hipMemcpyToSymbol(HIP_SYMBOL(Delta_d), Delta, sizeof(float));

  CompDataConn(NumInletNodes, MaxInletCoordY,
               MinInletCoordY, BCconn0, BCconn1, BCconn2, BCconn3, BCconn4,
               BCconn5, BCconn6, NumConn, Delta);

  hipMemcpy(Nodes0_d, Nodes0, (*NumNodes)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Nodes1_d, Nodes1, (*NumNodes)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Nodes2_d, Nodes2, (*NumNodes)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Nodes3_d, Nodes3, (*NumNodes)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Nodes4_d, Nodes4, (*NumNodes)*sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(BCconn0_d, BCconn0, (*NumConn)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BCconn1_d, BCconn1, (*NumConn)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BCconn2_d, BCconn2, (*NumConn)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BCconn3_d, BCconn3, (*NumConn)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BCconn4_d, BCconn4, (*NumConn)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(BCconn5_d, BCconn5, (*NumConn)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(BCconn6_d, BCconn6, (*NumConn)*sizeof(int), hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(InletProfile_d), &InletProfile, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(rho_ini_d), &rho_ini, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(Uavg_d), &Uavg, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(Vavg_d), &Vavg, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(MinInletCoordY_d), MinInletCoordY, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(MaxInletCoordY_d), MaxInletCoordY, sizeof(float));

  // Check whether we got back what we wanted :), wtite to log file!
  log_file = fopen(logFile, "w");  // open log file
  ppp      = &postproc_prog;       // for convenience ppp points to postproc_prog
  fprintf(log_file,"This is the 2D lattice Boltzmann *.log file\n\n");
  fprintf(log_file,"\n:::: Imported variables from the *.ini file :::: \n");
  fprintf(log_file,">>> Uavg              : %3.6f\n", Uavg);
  fprintf(log_file,">>> Vavg              : %3.6f\n", Vavg);
  fprintf(log_file,">>> Initial density   : %2.1f\n", rho_ini);
  fprintf(log_file,">>> Viscosity         : %3.8f\n", Viscosity);
  fprintf(log_file,">>> # of iterations   : %1.1d\n", Iterations);
  fprintf(log_file,">>> Autosave after    : %1.1d\n", AutosaveAfter);
  fprintf(log_file,">>> Autosave every    : %1.1d\n", AutosaveEvery);

  fprintf(log_file,">>> Convergence Veloc : UNKNOWN OPTION\n");
  fprintf(log_file,">>> Convergence Rho   : UNKNOWN OPTION\n");
  fprintf(log_file,">>> Former data iter# : UNKNOWN OPTION\n");


  switch(CollisionModel)         // 1: BGKW, 2: TRT, 3: MRT
  {
    case 1: fprintf(log_file,">>> CollisionModel   : BGKW\n"); break;
    case 2: fprintf(log_file,">>> CollisionModel   : TRT\n" ); break;
    case 3: fprintf(log_file,">>> CollisionModel   : MRT\n" ); break;
  }
  switch(InletProfile)                      // 1:ON, 2:OFF
  {
    case 1: fprintf(log_file,">>> InletProfile     : ON\n" ); break;
    case 2: fprintf(log_file,">>> InletProfile     : OFF\n"); break;
  }
  switch(OutletProfile)                     // 1:ON, 2:OFF
  {
    case 1: fprintf(log_file,">>> OutletProfile    : ON\n" ); break;
    case 2: fprintf(log_file,">>> OutletProfile    : OFF\n"); break;
  }
  switch(CurvedBoundaries)                  // 1:ON, 2:OFF
  {
    case 1: fprintf(log_file,">>> CurvedBoundaries : ON\n" ); break;
    case 2: fprintf(log_file,">>> CurvedBoundaries : OFF\n"); break;
  }
  switch(postproc_prog)   // 1->Paraview (*.csv)     2->Tecplot 
  {
    case 1: fprintf(log_file,">>> Results format   : Paraview (*.csv)\n" ); break;
    case 2: fprintf(log_file,">>> Results format   : Tecplot (*.dat)\n"); break;
  }
  if (CalculateDragLift != 0)
            fprintf(log_file,">>> Drag, lift @ BC  : %d\n", CalculateDragLift);
  else 
            fprintf(log_file,">>> Drag, lift was not calculated\n");


  fprintf(log_file,"\n:::: Calculated variables from mesh :::: \n");
  fprintf(log_file,">>> Grid spacing        = %f\n", *Delta);
  fprintf(log_file,">>> # of nodes in x (n) = %d\n", *n);
  fprintf(log_file,">>> # of nodes in y (m) = %d\n", *m);
  fprintf(log_file,">>> NumInletNodes       = %d\n", *NumInletNodes);
  fprintf(log_file,">>> MaxInletCoordY      = %f\n", *MaxInletCoordY);
  fprintf(log_file,">>> MinInletCoordY      = %f\n", *MinInletCoordY);
  // End of checking

  ////////////////////////////////////////////////////
  ///////////////// Allocate GPU /////////////////////
  ////////////////////////////////////////////////////

  // host scalar
  omega_h = Create1DArrayFloat(1);
  omegaA_h = Create1DArrayFloat(1);
  OutletProfile_h = Create1DArrayInt(1);
  CurvedBoundaries_h = Create1DArrayInt(1);
  CalculateDragLift_h = Create1DArrayInt(1);
  

  // declare and allocate 1D constant gpu and cpu vectors
  int *Fluid, *Corner, *ID, *Boundary, *BoundaryID;
  float *CoordX, *CoordY;

  Fluid = Create1DArrayInt((*n)*(*m));
  Corner = Create1DArrayInt((*n)*(*m));          
  ID = Create1DArrayInt((*n)*(*m));             
  Boundary = Create1DArrayInt((*n)*(*m));    
  BoundaryID = Create1DArrayInt((*n)*(*m));  
  CoordX = Create1DArrayFloat((*n)*(*m));   
  CoordY = Create1DArrayFloat((*n)*(*m));

  int *Fluid_d, *Corner_d, *ID_d, *Boundary_d, *BoundaryID_d;
  float *CoordX_d, *CoordY_d;

  hipMalloc((void**)&Fluid_d, (*m)*(*n) * sizeof(int));
  hipMalloc((void**)&Corner_d, (*m)*(*n) * sizeof(int));
  hipMalloc((void**)&ID_d, (*m)*(*n) * sizeof(int));
  hipMalloc((void**)&Boundary_d, (*m)*(*n) * sizeof(int));
  hipMalloc((void**)&BoundaryID_d, (*m)*(*n) * sizeof(int));
  hipMalloc((void**)&CoordX_d, (*m)*(*n) * sizeof(float));
  hipMalloc((void**)&CoordY_d, (*m)*(*n) * sizeof(float));

// declare and allocate 9D constant gpu and cpu vectors
  int *StreamLattice, *BC_ID;
  float *Q;  

  StreamLattice = Create1DArrayInt(9*(*n)*(*m));
  BC_ID = Create1DArrayInt(9*(*n)*(*m));          
  Q = Create1DArrayFloat(9*(*n)*(*m));             

  int *StreamLattice_d, *BC_ID_d;
  float *Q_d;

  hipMalloc((void**)&StreamLattice_d, 9*(*m)*(*n) * sizeof(int));
  hipMalloc((void**)&BC_ID_d, 9*(*m)*(*n) * sizeof(int));
  hipMalloc((void**)&Q_d, 9*(*m)*(*n) * sizeof(float));


 // declare and allocate 1D variable gpu and cpu vectors
  float *U, *V, *Rho, *Uo, *Vo, *DragF, *LiftF;

  U = Create1DArrayFloat((*n)*(*m));
  V = Create1DArrayFloat((*n)*(*m));
  Rho = Create1DArrayFloat((*n)*(*m));
  Uo = Create1DArrayFloat((*n)*(*m));
  Vo = Create1DArrayFloat((*n)*(*m));
  DragF = Create1DArrayFloat((*n)*(*m));
  LiftF = Create1DArrayFloat((*n)*(*m));


  float *U_d, *V_d, *Rho_d, *Uo_d, *Vo_d, *DragF_d, *LiftF_d;

  hipMalloc((void**)&U_d, (*m)*(*n) * sizeof(float));
  hipMalloc((void**)&V_d, (*m)*(*n) * sizeof(float));
  hipMalloc((void**)&Rho_d, (*m)*(*n) * sizeof(float));
  hipMalloc((void**)&Uo_d, (*m)*(*n) * sizeof(float));
  hipMalloc((void**)&Vo_d, (*m)*(*n) * sizeof(float));
  hipMalloc((void**)&DragF_d, (*m)*(*n) * sizeof(float));
  hipMalloc((void**)&LiftF_d, (*m)*(*n) * sizeof(float));


// declare and allocate 9D constant gpu and cpu vectors
  float *F, *Feq, *METAF, *Fneighbours, *fmom, *fmeq, *sumb;  
    
  F = Create1DArrayFloat(9*(*n)*(*m));
  Feq = Create1DArrayFloat(9*(*n)*(*m));  
  METAF = Create1DArrayFloat(9*(*n)*(*m));  
  Fneighbours = Create1DArrayFloat(9*(*n)*(*m));  
  fmom = Create1DArrayFloat(9*(*n)*(*m));  
  fmeq = Create1DArrayFloat(9*(*n)*(*m));  
  sumb = Create1DArrayFloat(9*(*n)*(*m));               

  float *F_d, *Feq_d, *METAF_d, *Fneighbours_d, *fmom_d, *fmeq_d, *sumb_d;

  hipMalloc((void**)&F_d, 9*(*m)*(*n) * sizeof(float));
  hipMalloc((void**)&Feq_d, 9*(*m)*(*n) * sizeof(float));
  hipMalloc((void**)&METAF_d, 9*(*m)*(*n) * sizeof(float));
  hipMalloc((void**)&Fneighbours_d, 9*(*m)*(*n) * sizeof(float));
  hipMalloc((void**)&fmom_d, 9*(*m)*(*n) * sizeof(float));
  hipMalloc((void**)&fmeq_d, 9*(*m)*(*n) * sizeof(float));
  hipMalloc((void**)&sumb_d, 9*(*m)*(*n) * sizeof(float));


  ////////////////////////////////////////////////////
  ///////////////// INITIALIZE ///////////////////////
  ////////////////////////////////////////////////////

  // In case of no autosave
  sprintf(AutosaveOutputFile, "NOWHERE!");


  //CONSTANT LATTICE QUANTITIES
  w[0]=4./9.;

  for (i=1; i<5; i++ )
      w[i]=1./9.;

  for (i=5; i<9; i++ )
      w[i]=1./36.;


  opp[0] = 0;
  opp[1] = 3*(*m)*(*n);
  opp[2] = 4*(*m)*(*n);
  opp[3] = 1*(*m)*(*n);
  opp[4] = 2*(*m)*(*n);
  opp[5] = 7*(*m)*(*n);
  opp[6] = 8*(*m)*(*n);
  opp[7] = 5*(*m)*(*n);
  opp[8] = 6*(*m)*(*n);

  cx[0] =  0;
  cx[1] =  1;
  cx[2] =  0;
  cx[3] = -1;
  cx[4] =  0;
  cx[5] =  1;
  cx[6] = -1;
  cx[7] = -1;
  cx[8] =  1;

  cy[0] =  0;
  cy[1] =  0;
  cy[2] =  1;
  cy[3] =  0;
  cy[4] = -1;
  cy[5] =  1;
  cy[6] =  1;
  cy[7] = -1;
  cy[8] = -1;

  c_h[0]=0;
  c_h[1]=-1;
  c_h[2]=-1*(*n);
  c_h[3]=1;
  c_h[4]=(*n);
  c_h[5]=-1*(*n)-1;
  c_h[6]=-1*(*n)+1;
  c_h[7]=(*n)+1;
  c_h[8]=(*n)-1;
  
  // Calculate collision freq
  Omega  = 1.0/(3.*Viscosity+0.5);
  OmegaA = 8*(2-Omega)/(8-Omega);

  omega_h=&Omega;
  omegaA_h=&OmegaA;
  OutletProfile_h=&OutletProfile;
  CalculateDragLift_h=&CalculateDragLift;
  CurvedBoundaries_h=&CurvedBoundaries;

  ////////////////////////////////////////////////////
  ///////////// COPY CONSTANTS TO GPU ////////////////
  ////////////////////////////////////////////////////

  hipMemcpyToSymbol(HIP_SYMBOL(cx_d), cx, 9*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cy_d), cy, 9*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(c_d), c_h, 9*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(opp_d), opp, 9*sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL(OutletProfile_d), OutletProfile_h, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(CurvedBoundaries_d), CurvedBoundaries_h, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(CalculateDragLift_d), CalculateDragLift_h, sizeof(int));

  hipMemcpyToSymbol(HIP_SYMBOL(width_d), m, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(height_d), n, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(Qlat_d), Qlat, 9*sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(w_d), w, 9*sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(omega_d), omega_h, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(omegaA_d), omegaA_h, sizeof(float));

  // Initialize variables for MRT Collision model, if used
  if (CollisionModel == 3)
  {
    float *tm, *stmiv;    // variables for the MRT collision model
    tm    = Create1DArrayFloat(81);
    stmiv = Create1DArrayFloat(81);
    MRTInitializer(tm, stmiv, Omega);

    hipMemcpyToSymbol(HIP_SYMBOL(tm_d), tm, 81*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(stmiv_d), stmiv, 81*sizeof(float));

  }



  // Create structure for the cell properties (see ShellFunctions.h)

  bx=(int)(9*(*m)*(*n)/threads)+1;

  dim3 tpb(threads); // threads/block
  dim3 bpg(bx); // blocks/grid

  // initializing matrix of struct Cells
  fprintf(log_file,"\n:::: Initializing ::::\n");
  printf("\n:::: Initializing ::::\n");
  tInstant1 = clock(); // Measure time of initialization
  

  ////////////////////////////////////////////////////
  ///////////// COPY VARIABLES TO GPU ////////////////
  ////////////////////////////////////////////////////
  
/*
  hipMemcpy(Fluid_d, Fluid, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Corner_d, Corner, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ID_d, ID, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Boundary_d, Boundary, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BoundaryID_d, BoundaryID, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(CoordX_d, CoordX, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(CoordY_d, CoordY, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(StreamLattice_d, StreamLattice, 9*(*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BC_ID_d, BC_ID, 9*(*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Q_d, Q, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(U_d, U, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(V_d, V, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Rho_d, Rho, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Uo_d, Uo, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Vo_d, Vo, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(DragF_d, DragF, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(LiftF_d, LiftF, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(F_d, F_d, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Feq_d, Feq, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(METAF_d, METAF, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Fneighbours_d, Fneighbours, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(fmom_d, fmom, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(fmeq_d, fmeq, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(sumb_d, sumb, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
*/

/*
  hipMemcpy(Fluid_d, Fluid, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ID_d, ID, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Boundary_d, Boundary, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BoundaryID_d, BoundaryID, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(CoordX_d, CoordX, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(CoordY_d, CoordY, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(BC_ID_d, BC_ID, 9*(*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Q_d, Q, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(U_d, U, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(V_d, V, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Rho_d, Rho, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
*/


  hipMemcpy(Fluid_d, Fluid, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Corner_d, Corner, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ID_d, ID, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Boundary_d, Boundary, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BoundaryID_d, BoundaryID, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(CoordX_d, CoordX, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(CoordY_d, CoordY, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(StreamLattice_d, StreamLattice, 9*(*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BC_ID_d, BC_ID, 9*(*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Q_d, Q, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(U_d, U, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(V_d, V, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Rho_d, Rho, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Uo_d, Uo, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Vo_d, Vo, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(DragF_d, DragF, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(LiftF_d, LiftF, (*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(F_d, F, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Feq_d, Feq, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(METAF_d, METAF, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(Fneighbours_d, Fneighbours, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);

switch(CollisionModel)
    {
      
      case 1:
      break;
      
      case 2:
      break;
      
      case 3:
          hipMemcpy(fmom_d, fmom, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
          hipMemcpy(fmeq_d, fmeq, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
          hipMemcpy(sumb_d, sumb, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);
      break;
      
    }




  gpu_init1<<<bpg,tpb>>>(Fluid_d, ID_d, Boundary_d, BoundaryID_d, CoordX_d, CoordY_d, BC_ID_d, Q_d, U_d, V_d, Rho_d,
                        Nodes0_d, Nodes1_d, Nodes2_d, Nodes3_d, Nodes4_d, BCconn0_d,
                        BCconn1_d, BCconn2_d, BCconn3_d, BCconn4_d, BCconn5_d, BCconn6_d);

  hipMemcpy(BC_ID, BC_ID_d, 9*(*m)*(*n)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(BoundaryID, BoundaryID_d, (*m)*(*n)*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(Q, Q_d, 9*(*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(CoordX, CoordX_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(CoordY, CoordY_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);


  for(B=0;B<*m;B++)
  {
  for(A=0;A<*n;A++)
  {
    ind=A+B*(*n);

    for(i=0;i<*NumConn;i++)
    {
      if ( ( *(BCconn0+i) == A ) && ( *(BCconn1+i) == B ) )
      {
          for(j=1; j<9;j++)
          {
              if ( *(BCconn2+i) == j )
              {
                ind9=A+B*(*n)+(*m)*(*n)*j;
                *(BC_ID + ind9)   = *(BCconn3+i);
                *(BoundaryID + ind) = *(BCconn6+i);

                // find distance from the boundary
                *(Q + ind9) = sqrt(pow( *(BCconn4+i)-*(CoordX+ind),2 ) + pow( *(BCconn5+i)-*(CoordY+ind),2) ) / ((*Delta)*Qlat[j]);
              }
          }
      }
    }
  }
  }  

  hipMemcpy(BC_ID_d, BC_ID, 9*(*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(BoundaryID_d, BoundaryID, (*m)*(*n)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Q_d, Q, 9*(*m)*(*n)*sizeof(float), hipMemcpyHostToDevice);


  gpu_init2<<<bpg,tpb>>>(Corner_d, Boundary_d, CoordY_d, StreamLattice_d, BC_ID_d, U_d, V_d, Uo_d, Vo_d,
                        Nodes0_d, Nodes1_d, Nodes2_d, Nodes3_d, Nodes4_d,
                        BCconn0_d, BCconn1_d, BCconn2_d, BCconn3_d, BCconn4_d,
                        BCconn5_d, BCconn6_d );

  hipMemcpy(U, U_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(V, V_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Rho, Rho_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Fluid, Fluid_d, (*m)*(*n)*sizeof(int), hipMemcpyDeviceToHost);


  tInstant2 = clock(); // Measure time of initialization
  tInitialization = (float)(tInstant2-tInstant1) / CLOCKS_PER_SEC;

  fprintf(log_file,"\n:::: Initialization done! ::::\n");

  printf("Initialization took %f seconds\n",         tInitialization);


  // Write Initialized data 
  switch(postproc_prog)
    {
      case 1: sprintf(OutputFile, "Results/InitialData.csv");   break;
      case 2: sprintf(OutputFile, "Results/InitialData.dat");   break;
    }

  //WriteResults(OutputFile, Cells_const_h, Cells_const_9d_h, Cells_var_h, Cells_var_9d_h, n, m, ppp);
  WriteResults(OutputFile, CoordX, CoordY, U, V, Rho, Fluid, n, m, ppp);
  
  printf("\nInitialized data was written to %s\n", OutputFile);

  // Open residuals file
  resid_file = fopen("Results/residuals.dat", "w");
  fprintf(resid_file,"Iter L2_norm L2_norm_weighted Drag Lift\n");


  ////////////////////////////////////////////////////
  /////////////////// ITERATE ////////////////////////
  ////////////////////////////////////////////////////

  fprintf(log_file,"\n:::: Start Iterations ::::\n");
  printf("\n:::: Start Iterations ::::\n");

  printf("%d is the number of iterations \n",         Iterations);


  tIterStart = clock(); // Start measuring time of main loop
  while (iter<Iterations)
  {
    
    ////////////// COLLISION ///////////////
    hipEventRecord(start, 0); // Start measuring time

    switch(CollisionModel)
    {
      
      case 1:
        gpu_bgk<<<bpg,tpb>>>(Fluid_d, Feq_d, Rho_d, U_d, V_d, METAF_d, F_d);
      break;
      
      case 2:
        gpu_trt1<<<bpg,tpb>>>(Fluid_d, Feq_d, Rho_d, U_d, V_d);
        gpu_trt2<<<bpg,tpb>>>(Fluid_d, Feq_d, F_d, METAF_d);
      break;
      
      case 3:
        gpu_mrt1<<<bpg,tpb>>>(Fluid_d, Rho_d, U_d, V_d, F_d, fmeq_d, fmom_d);
        gpu_mrt2<<<bpg,tpb>>>(Fluid_d, sumb_d, fmom_d, fmeq_d, METAF_d, F_d);
      break;
      
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cudatime, start, stop);
    tCollision = tCollision + cudatime;


    ////////////// UPDATE DISTR ///////////////
  	hipEventRecord(start, 0);

    gpu_update_f<<<bpg,tpb>>>(Fluid_d, F_d, METAF_d);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cudatime, start, stop);
    tUpdateF = tUpdateF + cudatime;

    
    ////////////// STREAMING ///////////////
    hipEventRecord(start, 0);

    gpu_streaming<<<bpg,tpb>>>(Fluid_d, StreamLattice_d, F_d, METAF_d);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cudatime, start, stop);
    tStreaming = tStreaming + cudatime;

  // make the host block until the device is finished with foo
  hipDeviceSynchronize();

  // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

    ////////////// BOUNDARIES ///////////////
    hipEventRecord(start, 0);

    gpu_boundaries1<<<bpg,tpb>>>(Fluid_d, Boundary_d, BC_ID_d, F_d, Uo_d, Vo_d, Corner_d);

    gpu_boundaries2<<<bpg,tpb>>>(Fluid_d, Fneighbours_d, METAF_d, BC_ID_d, Q_d, F_d);
    gpu_boundaries3<<<bpg,tpb>>>(Fluid_d, BC_ID_d, F_d, Uo_d, Vo_d, Corner_d);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cudatime, start, stop);
    tBoundaries = tBoundaries + cudatime;


    // UPDATE VELOCITY AND DENSITY
    hipEventRecord(start, 0);

    gpu_update_macro<<<bpg,tpb>>>(Fluid_d, Rho_d, U_d, V_d, BC_ID_d, BoundaryID_d, DragF_d, LiftF_d, CoordX_d, CoordY_d, F_d);

    tInstant2 = clock();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cudatime, start, stop);
    tUpdateMacro = tUpdateMacro + cudatime;
    

    ////////////////////////////////////////////////////
    //////////// COPY VARIABLES TO HOST ////////////////
    ////////////////////////////////////////////////////
/*
    hipMemcpy(F, F_d, (*m)*(*n)*9*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(METAF, METAF_d, (*m)*(*n)*9*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(DragF, DragF_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(LiftF, LiftF_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
*/
    ////////////// Residuals ///////////////
/*
    tInstant1 = clock(); // Start measuring time
    
    ComputeResiduals(BoundaryID, F, METAF, DragF, LiftF, Residuals, m, n, CalculateDragLift);
    fprintf(resid_file,"%d %5.4e %5.4e %f %f\n", iter, Residuals[0], Residuals[1], Residuals[2], Residuals[3]);
    
    tInstant2 = clock();
    tResiduals = tResiduals + (float)(tInstant2-tInstant1) / CLOCKS_PER_SEC;
*/
    printf("Iterating... %d/%d (%3.1f %%)\r", iter+1, Iterations, (float)(iter+1)*100/(float)(Iterations));

    // update loop variable
    iter++;  
   
    ////////////// Autosave ///////////////
    
    if(iter == (AutosaveEvery*AutosaveI))
    {
      AutosaveI++;
      if(iter>AutosaveAfter)
      {

        ////////////////////////////////////////////////////
        //////////// COPY VARIABLES TO HOST ////////////////
        ////////////////////////////////////////////////////

        hipMemcpy(U, U_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(V, V_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(Rho, Rho_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(Fluid, Fluid_d, (*m)*(*n)*sizeof(int), hipMemcpyDeviceToHost);
  
        switch(postproc_prog)
        {
          case 1: sprintf(AutosaveOutputFile, "Results/autosave_iter%05d.csv", iter);
          break;
          case 2: sprintf(AutosaveOutputFile, "Results/autosave_iter%05d.dat", iter);
          break;
        }

      tInstant1 = clock(); // Start measuring time
      WriteResults(AutosaveOutputFile, CoordX, CoordY, U, V, Rho, Fluid, n, m, ppp);
      tInstant2 = clock();
      tWriting = tWriting + (float)(tInstant2-tInstant1) / CLOCKS_PER_SEC;
      }
    }
  

  }     ////////////// END OF MAIN WHILE CYCLE! ///////////////

  tIterEnd = clock(); // End measuring time of main loop
  tIteration = (float)(tIterEnd - tIterStart ) / CLOCKS_PER_SEC;

  ////////////////////////////////////////////////////
  //////////// COPY VARIABLES TO HOST ////////////////
  ////////////////////////////////////////////////////
  hipMemcpy(U, U_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(V, V_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Rho, Rho_d, (*m)*(*n)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Fluid, Fluid_d, (*m)*(*n)*sizeof(int), hipMemcpyDeviceToHost);

  clock_t tEnd = clock();
  float tOverall = (float)(tEnd - tStart) / CLOCKS_PER_SEC; // Calculate elapsed time

  fprintf(log_file,"\nOverall calculations took %f seconds", tOverall);
  fprintf(log_file,"\nMain while loop took %f seconds\n",      tIteration);
  fprintf(log_file,"Initialization took %f seconds\n",         tInitialization);
  fprintf(log_file,"Collision took %f seconds\n",              tCollision/1000);
  fprintf(log_file,"UpdateF took %f seconds\n",                tUpdateF/1000);
  fprintf(log_file,"Streaming took %f seconds\n",              tStreaming/1000);
  fprintf(log_file,"Calculating Boundaries took %f seconds\n", tBoundaries/1000);
  fprintf(log_file,"Update Macroscopic took %f seconds\n",     tUpdateMacro/1000);
  fprintf(log_file,"Calculating Residuals took %f seconds\n",  tResiduals);
  fprintf(log_file,"Writing results took %f seconds\n",        tWriting);

  // end time measurement, close log file
  fprintf(log_file,"\n:::: Iterations done! ::::\n");
  fclose(log_file);

  // Close residuals file
  fclose(resid_file);
  
  // Write the time measurements to a separate dat file
  TimeMeasurementFile = fopen("Results/SerialTimeMeasuerment.dat","w");
  fprintf(TimeMeasurementFile,"tOverall %f\n",        tOverall);
  fprintf(TimeMeasurementFile,"tIteration %f\n",      tIteration);
  fprintf(TimeMeasurementFile,"tInitialization %f\n", tInitialization);
  fprintf(TimeMeasurementFile,"tCollision %f\n",      tCollision/1000);
  fprintf(TimeMeasurementFile,"tUpdateF %f\n",        tUpdateF/1000);
  fprintf(TimeMeasurementFile,"tStreaming %f\n",      tStreaming/1000);
  fprintf(TimeMeasurementFile,"tBoundaries %f\n",     tBoundaries/1000);
  fprintf(TimeMeasurementFile,"tUpdateMacro %f\n",    tUpdateMacro/1000);
  fprintf(TimeMeasurementFile,"tResiduals %f\n",      tResiduals);
  fprintf(TimeMeasurementFile,"tWriting %f\n",        tWriting);
   fclose(TimeMeasurementFile);
  
  // Write final data
  switch(postproc_prog)
  {
    case 1: sprintf(FinalOutputFile, "Results/FinalData.csv"); break;
    case 2: sprintf(FinalOutputFile, "Results/FinalData.dat"); break;
  }
  WriteResults(FinalOutputFile, CoordX, CoordY, U, V, Rho, Fluid, n, m, ppp);

  // Write information for user
  printf("\n\nLog was written to %s\n", logFile);
  printf("Last autosave result can be found at %s\n", AutosaveOutputFile);
  printf("Residuals were written to Results/residuals.dat\n");
  printf("Profiling results were written to Results/SerialTimeMeasuerment.dat\n");
  printf("Final results were written to %s\n", FinalOutputFile);

  ////////////////////////////////////////////////////
  ///////////////// End of line //////////////////////
  ////////////////////////////////////////////////////

  // FREE POINTERS
  free(Delta);
  free(m);
  free(n);
  free(MaxInletCoordY);
  free(MinInletCoordY);
  free(NumInletNodes);
  free(NumNodes);
  free(NumConn);
  free(w);
  free(cx);
  free(cy);
  free(opp);
  free(Residuals);

  hipFree (cx_d);
  hipFree (cy_d);
  hipFree (w_d);
  hipFree (omega_d);
  //...

}
