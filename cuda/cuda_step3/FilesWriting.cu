#include "hip/hip_runtime.h"
#include <stdio.h>   // for calloc();
#include "FilesWriting.h"
#include <math.h>

void WriteResults(char* OutputFile, FLOAT_TYPE* CoordX, FLOAT_TYPE* CoordY, FLOAT_TYPE* U,
                  FLOAT_TYPE* V, FLOAT_TYPE* Rho, int* Fluid, int n, int m, OutputFormat outputFormat)
{
  int i;                      // Loop variables
  FILE * fp1;                 // file pointer to output file
  fp1 = fopen(OutputFile, "w"); // open file
  switch(outputFormat)
  {
    case PARAVIEW:

      fprintf(fp1, "x,y,u,v,vel_mag,rho,press,fluid\n");
      for(i=0; i<m*n; ++i)
      {

        fprintf(fp1, "%f, %f, %f, %f, %f, %f, %f, %d\n",
                CoordX[i], // x
                CoordY[i], // y
                U[i],      // u
                V[i],      // v
                sqrt(pow(U[i],2)+pow(V[i],2)), // u magnitude
                Rho[i],    // density
                Rho[i]/3,  // pressure
                Fluid[i]);
      }
      fclose(fp1);
    break;

  	case TECPLOT:
    	fprintf(fp1, "Title = \"LBM results\"\n");
    	fprintf(fp1, "Variables = \"x\",\"y\",\"u\",\"v\",\"vel mag\",\"rho\",\"press\",\"fluid\"\n");
    	fprintf(fp1, "Zone i=%d, j=%d, f=point\n",n,m);

    	for(i=0; i<m*n; ++i)
    	{

        fprintf(fp1, "%f %f %f %f %f %f %f %d\n",
                CoordX[i], // x
                CoordY[i], // y
                U[i],      // u
                V[i],      // v
                sqrt(pow(U[i],2)+pow(V[i],2)), // u magnitude
                Rho[i],    // density
                Rho[i]/3,  // pressure
                Fluid[i]);
     	}

      fclose(fp1);
  	break;
  }


}
